#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

// Here, we do the operation in-place.
__global__ void scan_kernel_upsweep(int N, int two_d, int two_dplus1, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        result[(index + 1) * two_dplus1 - 1] += result[index * two_dplus1 + two_d - 1];
    }
}

// Here, we do the operation in-place.
__global__ void scan_kernel_downsweep(int N, int two_d, int two_dplus1, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        int temp = result[index * two_dplus1 + two_d - 1];
        result[index * two_dplus1 + two_d - 1] = result[(index + 1) * two_dplus1 - 1];
        result[(index + 1) * two_dplus1 - 1] += temp;
    }
}

// It may seem ugly, but it works.
__global__ void scan_set_zero_kernel(int N, int *result) {
    result[N - 1] = 0;
}

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int N, int* result)
{

    int rounded_length = nextPow2(N);

    for(int two_d = 1; two_d <= rounded_length / 2; two_d *= 2) {
        int two_dplus1= 2 * two_d;
        // Here, we should calculate the total task number we need
        int totalTaskNum = rounded_length % two_dplus1 ? rounded_length / two_dplus1 + 1 
                                                       : rounded_length / two_dplus1;
        // Thus, we could calculate the block we need.
        int blocks = totalTaskNum % THREADS_PER_BLOCK ? totalTaskNum / THREADS_PER_BLOCK + 1
                                                      : totalTaskNum / THREADS_PER_BLOCK;
        scan_kernel_upsweep<<<blocks, THREADS_PER_BLOCK>>>(totalTaskNum, two_d, two_dplus1, result);
        hipDeviceSynchronize();
    }

    // Here, I don't know how to find a good way
    // to set the result[N - 1] = 0. So I use
    // a simple way.
    scan_set_zero_kernel<<<1,1>>>(rounded_length, result);
    hipDeviceSynchronize();

    for(int two_d = rounded_length / 2; two_d >= 1; two_d /= 2) {
        int two_dplus1= 2 * two_d;
        int totalTaskNum = rounded_length % two_dplus1 ? rounded_length / two_dplus1 + 1 
                                                       : rounded_length / two_dplus1;
        int blocks = totalTaskNum % THREADS_PER_BLOCK ? totalTaskNum / THREADS_PER_BLOCK + 1
                                                      : totalTaskNum / THREADS_PER_BLOCK;
        scan_kernel_downsweep<<<blocks, THREADS_PER_BLOCK>>>(totalTaskNum, two_d, two_dplus1, result);
        hipDeviceSynchronize();
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);

    // Here I choose the in-place way
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration;
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void find_repeats_compare(int *input, int *output, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1) {
        output[index] = input[index] == input[index + 1] ? 1 : 0;
    }
}

__global__ void gather_kernel(int* exclusive_scan_results, int* output, const int N, int* total_count) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N - 1) {
        if (exclusive_scan_results[index] != exclusive_scan_results[index + 1]) {
            output[exclusive_scan_results[index]] = index;
        }
    } else if (index == N - 1) {
        *total_count = exclusive_scan_results[N - 1];
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // The problem is that how should we use `prefix_sum` to solve
    // the `find_repeats` problem. It is tricky, you should look at
    // the code carefully.

    const int rounded_length = nextPow2(length);
    int blocks = rounded_length % THREADS_PER_BLOCK ? rounded_length / THREADS_PER_BLOCK + 1
                                                    : rounded_length / THREADS_PER_BLOCK;

    int* temp;
    hipMalloc(&temp, rounded_length * sizeof(int));
    find_repeats_compare<<<blocks, THREADS_PER_BLOCK>>>(device_input, temp, length);

    // exclusive scan on indicator array
    // to get device_exclusive_scan_results
    exclusive_scan(length, temp);

    // get repetition points in array    
    int* device_repetition_count;
    hipMalloc(&device_repetition_count, sizeof(int));
    gather_kernel<<<blocks, THREADS_PER_BLOCK>>>(temp, device_output, length, device_repetition_count);
    hipFree(temp);

    // return results
    int output_length;
    hipMemcpy(&output_length, device_repetition_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_repetition_count);
    return output_length;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
